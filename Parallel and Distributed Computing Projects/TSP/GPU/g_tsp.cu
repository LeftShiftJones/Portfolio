/**
 * GPU Parallel Assignment
 * @author: Ryan Jones
 * Completed Sunday, December 2nd, 2018
 */

// includes, system

#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <unistd.h>
#include <limits.h>
#include <time.h>

#define N 1000

/* Reference an element in the TSP distance array. */
#define TSP_ELT(tsp, n, i, j) *(tsp + (i * n) + j)
#define ONE_BILLION (double)1000000000.0

/* Action function for each permuation. */
typedef void (*perm_action_t)(int *v, int n);


/////////////////////////////////////////////////////////////////////
// Kth Perm all the way down...
/////////////////////////////////////////////////////////////////////


typedef struct {
    int *values;  /* Values stored in list */
    int max_size; /* Maximum size allocated */
    int cur_size; /* Size currently in use */
} list_t;

/* Swap v[i] and v[j] */
__device__ void swap(int *v, int i, int j) {
    int t = v[i];
    v[i] = v[j];
    v[j] = t;
}

/* Dump list, including sizes */
__device__ void list_dump(list_t *list) {
    //printf("%2d/%2d", list->cur_size, list->max_size);
    for (int i = 0; i < list->cur_size; i++) {
        printf(" %d", list->values[i]);
    }
    printf("\n");
}

/* Allocate list that can store up to 'max_size' elements */
__device__ list_t *
list_alloc(int max_size) {
    list_t *list = (list_t *)malloc(sizeof(list_t));
    list->values = (int *)malloc(max_size * sizeof(int));
    list->max_size = max_size;
    list->cur_size = 0;
    return list;
}

/* Free a list; call this to avoid leaking memory! */
__device__ void list_free(list_t *list) {
    free(list->values);
    free(list);
}

/* Add a value to the end of the list */
__device__ void list_add(list_t *list, int value) {
    if (list->cur_size >= list->max_size) {
        printf("List full");
        list_dump(list);
        //exit(1);
    }
    list->values[list->cur_size++] = value;
}

/* Return the current size of the list */
__device__ int list_size(list_t *list) {
    return list->cur_size;
}

/* Validate index */
__device__ void _list_check_index(list_t *list, int index) {
    if (index < 0 || index > list->cur_size - 1) {
        printf("Invalid index %d\n", index);
        list_dump(list);
        //exit(1);
    }
}

/* Get the value at given index */
__device__ int list_get(list_t *list, int index) {
    _list_check_index(list, index);
    return list->values[index];
}

/* Remove the value at the given index */
__device__ void list_remove_at(list_t *list, int index) {
    _list_check_index(list, index);
    for (int i = index; i < list->cur_size - 1; i++) {
        list->values[i] = list->values[i + 1];
    }
    list->cur_size--;
}

/* Retrieve a copy of the values as a simple array of integers. The returned
   array is allocated dynamically; the caller must free the space when no
   longer needed.
 */
__device__ int *list_as_array(list_t *list) {
    int *rtn = (int *)malloc(list->max_size * sizeof(int));
    for (int i = 0; i < list->max_size; i++) {
        rtn[i] = list_get(list, i);
    }
    return rtn;
}
 

/* Calculate n! iteratively */
__device__ long factorial(int n) {
    if (n < 1) {
        return 0;
    }

    long rtn = 1;
    for (int i = 1; i <= n; i++) {
        rtn *= i;
    }
    return rtn;
}

long host_factorial(int n) {
    if (n < 1) {
        return 0;
    }

    long rtn = 1;
    for (int i = 1; i <= n; i++) {
        rtn *= i;
    }
    return rtn;
}

/* Return the kth lexographically ordered permuation of an array of size integers
   in the range [0 .. size - 1]. The integers are allocated dynamically and
   should be free'd by the caller when no longer needed.
*/
__device__ int *kth_perm(int k, int size) {
    long remain = k - 1;
    list_t *numbers = list_alloc(size);
    for (int i = 0; i < size; i++) {
        list_add(numbers, i);
    }

    list_t *perm = list_alloc(size);

    for (int i = 1; i < size; i++) {
        long f = factorial(size - i);
        long j = remain / f;
        remain = remain % f;

        list_add(perm, list_get(numbers, j));
        list_remove_at(numbers, j);

        if (remain == 0) {
            break;
        }
    }

    /* Append remaining digits */
    for (int i = 0; i < list_size(numbers); i++) {
        list_add(perm, list_get(numbers, i));
    }

    int *rtn = list_as_array(perm);
    list_free(perm);

    return rtn;
}


/* Given an array of size elements at perm, update the array in place to
   contain the lexographically next permutation. It is originally due to
   Dijkstra. The present version is discussed at:
   http://www.cut-the-knot.org/do_you_know/AllPerm.shtml
 */
__device__ void next_perm(int *perm, int size) {
    int i = size - 1;
    while (perm[i - 1] >= perm[i]) {
        i = i - 1;
    }

    int j = size;
    while (perm[j - 1] <= perm[i - 1]) {
        j = j - 1;
    }

    swap(perm, i - 1, j - 1);

    i++;
    j = size;
    while (i < j) {
        swap(perm, i - 1, j - 1);
        i++;
        j--;
    }
}


/////////////////////////////////////////////////////////////////////
// TSP all the way down...
/////////////////////////////////////////////////////////////////////

/* Create an instance of a symmetric TSP. */
int *create_tsp(int n, int seed) {
    int *tsp = (int *)malloc(n * n * sizeof(int));

    srandom(seed);
    for (int i = 0; i < n; i++) {
        for (int j = 0; j <= i; j++) {
            int val = (int)(random() / (RAND_MAX / 100));
            TSP_ELT(tsp, n, i, j) = val;
            TSP_ELT(tsp, n, j, i) = val;
        }
    }
    return tsp;
}

/* Evaluate a single instance of the TSP. */
__device__ int eval_tsp(int *perm, int n, int *distances) {
    /* Calculate the length of the tour for the current permutation. */
    int total = 0;
    for (int i = 0; i < n; i++) {
        int j = (i + 1) % n;
        int from = perm[i];
        int to = perm[j];
        int val = TSP_ELT(distances, n, from, to);
        total += val;
    }
    return total;
}

/* Print a TSP distance matrix. */
void print_tsp(int *tsp, int n, int random_seed) {
    printf("TSP (%d cities - seed %d)\n    ", n, random_seed);
    for (int j = 0; j < n; j++) {
        printf("%3d|", j);
    }
    printf("\n");
    for (int i = 0; i < n; i++) {
        printf("%2d|", i);
        for (int j = 0; j < n; j++) {
            printf("%4d", TSP_ELT(tsp, n, i, j));
        }
        printf("\n");
    }
    printf("\n");
}

/////////////////////////////////////////////////////////////////////
// My stuff all the way down...
/////////////////////////////////////////////////////////////////////

/**
 * 
 */
__global__ void compute_shortest_path(int num_cities, int num_threads, int *dists, int *mins) {
    int shortest_length = INT_MAX;
    int tid = threadIdx.x;
    long max_check = factorial(num_cities);
    long tours_to_check = max_check / num_threads;
    int *current = kth_perm((tid+1)*tours_to_check, num_cities);
    int at = tid*tours_to_check;
    do {
        int temp = eval_tsp(current, num_cities, dists);
        if(temp < shortest_length) {
            shortest_length = temp;
        }
        next_perm(current, num_cities);
        at++;
    } while (at < (int)((tid+1)*tours_to_check)-1);
    *(mins + (tid)) = shortest_length;
    free(current); 
}

//get current time
double now(void) {
    struct timespec current_time;
    clock_gettime(CLOCK_REALTIME, &current_time);
    return current_time.tv_sec + (current_time.tv_nsec / ONE_BILLION);
}

void usage(char *prog_name) {
    fprintf(stderr, "usage: %s [flags]\n", prog_name);
    fprintf(stderr, "   -h\n");
    fprintf(stderr, "   -c <number of cities>\n");
    fprintf(stderr, "   -s <random seed>\n");
    fprintf(stderr, "   -n <number of threads>\n");
    exit(1);
}

int main(int argc, char **argv) {
    int random_seed = 42;
    int num_cities = 5;
    int num_threads = 1;
    int ch;
    while ((ch = getopt(argc, argv, "c:hs:n:")) != -1) {
        switch (ch) {
        case 'c':
            num_cities = atoi(optarg);
            break;
        case 's':
            random_seed = atoi(optarg);
            break;
        case 'n':
            num_threads = atoi(optarg);
            break;
        case 'h':
        default:
            usage(argv[0]);
        }
    }
    long fact = host_factorial(num_cities);

    //Checks to see if factorial is essentially less than 1024
    //or the number of specified threads. If so, this statement
    //changes the number of threads to be a 1:1 ratio with the
    //permutations
    if((long)num_threads > fact) {
        num_threads = (int)fact;
        fprintf(stderr, "Too many processors to run effectively...\nRefactoring number of threads to be 1:1...\nNow running on %d GPU threads\n", num_threads);
    }
    
    //create host distances and minimum arrays
    int *h_distances = create_tsp(num_cities, random_seed);
    int *h_min_distances = (int *)malloc(num_threads * sizeof(int));

    //create device distances and minimum arrays
    int *d_distances; 
    int *d_min_distances;
    hipMalloc((void **)&d_distances, num_cities * num_cities * sizeof(int));
    hipMalloc((void **)&d_min_distances, num_threads * sizeof(int));

    hipMemcpy(d_distances, h_distances, num_cities * num_cities * sizeof(int), hipMemcpyHostToDevice);
    double start_time = now();
    compute_shortest_path<<<1, num_threads>>>(num_cities, num_threads, d_distances, d_min_distances);

    hipMemcpy(h_min_distances, d_min_distances, num_threads * sizeof(int), hipMemcpyDeviceToHost);

    hipFree(d_distances);
    hipFree(d_min_distances);
    
    int lowest = INT_MAX;
    for(int i = 0; i < num_threads; i++) {
        int val = *((h_min_distances) + i);
        if(val < lowest) {
            lowest = val;
        }
    }
    free(h_min_distances);
    free(h_distances);

    printf("Lowest tour %d found on seed %d in %5.3f seconds\n", lowest, random_seed, now() - start_time);
    
    return 0;
}
